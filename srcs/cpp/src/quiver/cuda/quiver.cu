#include <algorithm>
#include <numeric>

#include <thrust/device_vector.h>

#include <torch/extension.h>

#include <quiver/common.hpp>
#include <quiver/functor.cu.hpp>
#include <quiver/quiver.cu.hpp>
#include <quiver/reindex.cu.hpp>
#include <quiver/trace.hpp>
#include <quiver/zip.hpp>

namespace quiver
{
template <typename T>
void replicate_fill(size_t n, const T *counts, const T *values, T *outputs)
{
    for (size_t i = 0; i < n; ++i) {
        const size_t c = counts[i];
        std::fill(outputs, outputs + c, values[i]);
        outputs += c;
    }
}

using torch_quiver_t = quiver<int64_t, CUDA>;

class TorchQuiver : public torch_quiver_t
{
    using torch_quiver_t::torch_quiver_t;

  public:
    using T = int64_t;
    using W = float;

    // deprecated, not compatible with AliGraph
    std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
    sample_sub(const torch::Tensor &vertices, int k) const
    {
        return sample_sub_with_stream(0, vertices, k);
    }

    std::tuple<torch::Tensor, torch::Tensor>
    sample_once(const torch::Tensor &vertices, int k) const
    {
        TRACE(__func__);

        thrust::device_vector<T> inputs;
        thrust::device_vector<T> outputs;
        thrust::device_vector<T> output_counts;

        return sample_kernel(0, vertices, k, inputs, outputs, output_counts);
    }

    std::tuple<torch::Tensor, torch::Tensor>
    sample_kernel(const hipStream_t stream, const torch::Tensor &vertices,
                  int k, thrust::device_vector<T> &inputs,
                  thrust::device_vector<T> &outputs,
                  thrust::device_vector<T> &output_counts) const
    {
        T tot = 0;
        const auto policy = thrust::cuda::par.on(stream);
        thrust::device_vector<T> output_ptr;
        thrust::device_vector<T> output_eid;

        check_eq<long>(vertices.dim(), 1);
        const size_t bs = vertices.size(0);

        {
            TRACE("alloc_1");
            inputs.resize(bs);
            output_counts.resize(bs);
            output_ptr.resize(bs);
        }
        // output_ptr is exclusive prefix sum of output_counts(neighbor counts
        // <= k)
        {
            TRACE("prepare");
            thrust::copy(vertices.data_ptr<long>(),
                         vertices.data_ptr<long>() + bs, inputs.begin());
            this->degree(stream, inputs.data(), inputs.data() + inputs.size(),
                         output_counts.data());
            if (k >= 0) {
                thrust::transform(policy, output_counts.begin(),
                                  output_counts.end(), output_counts.begin(),
                                  cap_by<T>(k));
            }
            thrust::exclusive_scan(policy, output_counts.begin(),
                                   output_counts.end(), output_ptr.begin());
            tot = thrust::reduce(policy, output_counts.begin(),
                                 output_counts.end());
        }
        {
            TRACE("alloc_2");
            outputs.resize(tot);
            output_eid.resize(tot);
        }
        // outputs[outptr[i], outptr[i + 1]) are unique neighbors of inputs[i]
        {
            TRACE("sample");
            this->sample(stream, inputs.begin(), inputs.end(),
                         output_ptr.begin(), output_counts.begin(),
                         outputs.data(), output_eid.data());
        }
        torch::Tensor neighbor = torch::empty(tot, vertices.options());
        torch::Tensor eid = torch::empty(tot, vertices.options());
        thrust::copy(outputs.begin(), outputs.end(), neighbor.data_ptr<T>());
        thrust::copy(output_eid.begin(), output_eid.end(), eid.data_ptr<T>());
        return std::make_tuple(neighbor, eid);
    }

    std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
    sample_sub_with_stream(const hipStream_t stream,
                           const torch::Tensor &vertices, int k) const
    {
        TRACE(__func__);
        const auto policy = thrust::cuda::par.on(stream);
        const size_t bs = vertices.size(0);

        thrust::device_vector<T> inputs;
        thrust::device_vector<T> outputs;
        thrust::device_vector<T> output_counts;
        thrust::device_vector<T> subset;

        sample_kernel(stream, vertices, k, inputs, outputs, output_counts);
        T tot = outputs.size();

        // reindex
        {
            {
                TRACE("reindex 0");
                subset.resize(inputs.size() + outputs.size());
                thrust::copy(policy, inputs.begin(), inputs.end(),
                             subset.begin());
                thrust::copy(policy, outputs.begin(), outputs.end(),
                             subset.begin() + inputs.size());
                thrust::sort(policy, subset.begin(), subset.end());
                subset.erase(thrust::unique(subset.begin(), subset.end()),
                             subset.end());
                _reindex_with(policy, outputs, subset, outputs);
            }
            {
                TRACE("permute");
                thrust::device_vector<T> s1;
                s1.reserve(subset.size());
                _reindex_with(policy, inputs, subset, s1);
                complete_permutation(s1, subset.size());
                subset = permute(s1, subset);

                thrust::device_vector<T> s2;
                inverse_permutation(s1, s2);
                permute_value(s2, outputs);
            }

            torch::Tensor out_vertices =
                torch::empty(subset.size(), vertices.options());
            torch::Tensor row_idx = torch::empty(tot, vertices.options());
            torch::Tensor col_idx = torch::empty(tot, vertices.options());
            {
                TRACE("prepare output");
                std::vector<T> counts(output_counts.size());
                std::vector<T> seq(output_counts.size());
                thrust::copy(output_counts.begin(), output_counts.end(),
                             counts.begin());
                std::iota(seq.begin(), seq.end(), 0);

                replicate_fill(bs, counts.data(), seq.data(),
                               row_idx.data_ptr<T>());
                thrust::copy(subset.begin(), subset.end(),
                             out_vertices.data_ptr<T>());
                thrust::copy(outputs.begin(), outputs.end(),
                             col_idx.data_ptr<T>());
            }
            return std::make_tuple(out_vertices, row_idx, col_idx);
        }
    }
};

TorchQuiver new_quiver_from_edge_index(size_t n,  //
                                       const torch::Tensor &edges,
                                       const torch::Tensor &edge_idx)
{
    TRACE(__func__);
    using T = typename TorchQuiver::T;
    check(edges.is_contiguous());
    check_eq<int64_t>(edges.dim(), 2);
    check_eq<int64_t>(edges.size(0), 2);
    const size_t m = edges.size(1);
    check_eq<int64_t>(edge_idx.dim(), 1);
    check_eq<int64_t>(edge_idx.size(0), m);

    thrust::device_vector<T> row_idx(m);
    thrust::device_vector<T> col_idx(m);
    {
        const T *p = edges.data_ptr<T>();
        thrust::copy(p, p + m, row_idx.begin());
        thrust::copy(p + m, p + m * 2, col_idx.begin());
    }
    thrust::device_vector<T> edge_idx_(m);
    {
        const T *p = edge_idx.data_ptr<T>();
        thrust::copy(p, p + m, edge_idx_.begin());
    }
    return TorchQuiver(static_cast<T>(n), std::move(row_idx),
                       std::move(col_idx), std::move(edge_idx_));
}

TorchQuiver new_quiver_from_edge_index_weight(size_t n,
                                              const torch::Tensor &edges,
                                              const torch::Tensor &edge_idx,
                                              const torch::Tensor &edge_weight)
{
    TRACE(__func__);
    using T = typename TorchQuiver::T;
    using W = typename TorchQuiver::W;
    check(edges.is_contiguous());
    check_eq<int64_t>(edges.dim(), 2);
    check_eq<int64_t>(edges.size(0), 2);
    const size_t m = edges.size(1);
    check_eq<int64_t>(edge_idx.dim(), 1);
    check_eq<int64_t>(edge_idx.size(0), m);

    thrust::device_vector<T> row_idx(m);
    thrust::device_vector<T> col_idx(m);
    {
        const T *p = edges.data_ptr<T>();
        thrust::copy(p, p + m, row_idx.begin());
        thrust::copy(p + m, p + m * 2, col_idx.begin());
    }
    thrust::device_vector<T> edge_idx_(m);
    {
        const T *p = edge_idx.data_ptr<T>();
        thrust::copy(p, p + m, edge_idx_.begin());
    }
    thrust::device_vector<W> edge_weight_(m);
    {
        const T *p = edge_weight.data_ptr<T>();
        thrust::copy(p, p + m, edge_weight_.begin());
    }
    return TorchQuiver(static_cast<T>(n), std::move(row_idx),
                       std::move(col_idx), std::move(edge_idx_),
                       std::move(edge_weight_));
}
}  // namespace quiver

void register_cuda_quiver(pybind11::module &m)
{
    m.def("new_quiver_from_edge_index", &quiver::new_quiver_from_edge_index);
    m.def("new_quiver_from_edge_index_weight",
          &quiver::new_quiver_from_edge_index_weight);
    py::class_<quiver::TorchQuiver>(m, "Quiver")
        .def("sample_sub", &quiver::TorchQuiver::sample_sub)
        .def("sample", &quiver::TorchQuiver::sample_once);
}
