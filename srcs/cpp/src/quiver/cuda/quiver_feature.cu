#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <pybind11/numpy.h>
#include <quiver/common.hpp>
#include <quiver/quiver.cu.hpp>
#include <quiver/shard_tensor.cu.hpp>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/csrc/utils/python_numbers.h>
//#include <ATen/MapAllocator.h>
#include <atomic>
#include <string>
#include <iostream>

namespace quiver
{
#define CHECK_CPU(x)                                                          \
AT_ASSERTM(!x.device().is_cuda(), #x " must be CPU tensor")

bool can_device_access_peer(int src_device_index, int dst_device_index){
    if (src_device_index == dst_device_index) {
        return true;
    }
    int access_i_j = 0, access_j_i = 0;
    hipDeviceCanAccessPeer(&access_i_j, src_device_index, dst_device_index);
    hipDeviceCanAccessPeer(&access_j_i, dst_device_index, src_device_index);
    return (access_i_j == 1) && (access_j_i == 1);
}

class ShardTensorItem
{
  public:
    int device;
    hipIpcMemHandle_t mem_handle;
    std::vector<int> shape;
    // for now we assume it is all float
    int dtype;
    ShardTensorItem(int device_, hipIpcMemHandle_t mem_handle_, std::vector<int> shape_):device(device_), mem_handle(mem_handle_), shape(shape_)
    {

    }
    ShardTensorItem(){

    };
    std::tuple<int, py::bytes, std::vector<int>> share_ipc(){
        auto _handle = PyBytes_FromStringAndSize((char *)&mem_handle, HIP_IPC_HANDLE_SIZE);
        auto bytes_obj = py::reinterpret_steal<py::object>((PyObject*)_handle);
        return std::make_tuple(device, bytes_obj, shape);
    }
    void from_ipc(std::tuple<int, std::string, std::vector<int>> ipc_data){

        device = std::get<0>(ipc_data);
        shape = std::get<2>(ipc_data);
        auto handle = std::get<1>(ipc_data);
        auto ipc_handle = reinterpret_cast<const hipIpcMemHandle_t*>(handle.c_str());

        mem_handle = *ipc_handle;
    }


};

class ShardTensor
{
  public:
    ShardTensor(int device) : device_(device), inited_(false), device_count_(0)
    {

        offset_list_.push_back(0);
    }

    size_t get_tensor_bytes(torch::Tensor tensor){
        // assume it's float
        int dim = tensor.dim();
        size_t total_bytes = 4;
        for(int index = 0; index < dim; index++){
            total_bytes *= tensor.sizes()[index];
        }
        return total_bytes;
    }

    std::vector<int> get_tensor_shape(torch::Tensor tensor){
        std::vector<int> shape;
        int dim = tensor.dim();
        for(int index = 0; index < dim; index++){
            shape.push_back(tensor.sizes()[index]);
        }
        return shape;
    }

    void append(ShardTensorItem item){
        hipSetDevice(device_);
        if (!inited_) {
            shape_.resize(item.shape.size());
            shape_[0] = 0;
            auto tensor_sizes = item.shape;
            for (int index = 1; index < shape_.size(); index++) {
                shape_[index] = tensor_sizes[index];
            }
            inited_ = true;
        }
        offset_list_.push_back(offset_list_[offset_list_.size() - 1] + item.shape[0]);

        // Check accessbility
        if(item.device >= 0){
            // TODO
            access_book.push_back(can_device_access_peer(device_, item.device));
        }else{
            access_book.push_back(1);
            //printf("%d <-> CPU support peer access \n", device_);
        }
        // get dev_ptr that can be accessed from this process
        void *ptr = NULL;
        tensor_devices_.push_back(item.device);
        if(!access_book[access_book.size() - 1]){
            hipSetDevice(item.device);
            hipIpcOpenMemHandle(&ptr, item.mem_handle, hipIpcMemLazyEnablePeerAccess);
            hipSetDevice(device_);
            //printf("WARNING: Tensor from device %d can NOT be accessed in kernel launched on device %d \n", item.device, device_);
        }else{
            hipIpcOpenMemHandle(&ptr, item.mem_handle, hipIpcMemLazyEnablePeerAccess);
        }

        //
        dev_ptrs_.push_back((float*)ptr);
        shape_[0] += item.shape[0];
        device_count_ += 1;
        cudaCheckError();

    }

    void append(torch::Tensor &tensor, int target_device)
    {
        CHECK_CPU(tensor);
        // for now, we assume tensor is added ordered
        if (!inited_) {
            shape_.resize(tensor.dim());
            shape_[0] = 0;
            auto tensor_sizes = tensor.sizes();
            for (int index = 1; index < shape_.size(); index++) {
                shape_[index] = tensor_sizes[index];
            }
            inited_ = true;

        }
        tensor_shapes_.push_back(get_tensor_shape(tensor));

        offset_list_.push_back(offset_list_[offset_list_.size() - 1] + tensor.sizes()[0]);

        void *ptr = NULL;
        size_t data_size = get_tensor_bytes(tensor);
        tensor_devices_.push_back(target_device);
        if(target_device >= 0){
            // if target_device >= 0, it means we use p2p
            //printf("LOG >>> Malloc Data On Device %d With %ulld Bytes\n", target_device, data_size);
            hipSetDevice(target_device);
            hipMalloc(&ptr, data_size);
            hipMemcpy(ptr, tensor.data_ptr<float>(), data_size, hipMemcpyHostToDevice);
            hipSetDevice(device_);

            // decide access book
            access_book.push_back(can_device_access_peer(device_, target_device));
        }else{
            hipSetDevice(device_);
            // if target_device < 0, it means we use Zero-Copy
            hipHostRegister(tensor.data_ptr<float>(), data_size, hipHostRegisterMapped);
            hipHostGetDevicePointer(&ptr, (void *)tensor.data_ptr<float>(), 0);
            access_book.push_back(1);
            //printf("%d <-> CPU support peer access \n", device_);
        }

        dev_ptrs_.push_back((float*)ptr);

        shape_[0] += tensor.size(0);
        device_count_ += 1;

    }

    torch::Tensor operator[](torch::Tensor &indices)
    {
        /*
        __global__ void quiver_tensor_gather(const int64_t** dev_ptrs, const
        int64_t* offsets, const int device_count, const int64_t* indices, int
        indice_length, const float* res, const int item_byte_size){
        torch::zeros((100,100),torch::KF32);
        */
        int current_device = 0;
        hipGetDevice(&current_device);
        auto stream = at::cuda::getCurrentCUDAStream();

        std::vector<int64_t> res_shape(shape_);
        res_shape[0] = indices.numel();
        // decide Tensor
        auto options = torch::TensorOptions()
                           .dtype(at::kFloat)
                           .device(torch::kCUDA, current_device);
        auto res = torch::empty(res_shape, options);
        cudaCheckError();

        // Device Data
        //for(int index = 0; index < offset_list_.size(); index++){
        //    std::cout<<"offset " << offset_list_[index]<<std::endl;
        //    std::cout<<"access_book[index] " << access_book[index]<<std::endl;
        //}

        float **buffers_device;
        int64_t *offset_device;
        int *access_book_device;

        // Copy buffers Device
        hipMalloc((void ***)&buffers_device, sizeof(float *) * device_count_);
        hipMemcpy(buffers_device, &dev_ptrs_[0],
                sizeof(float *) * dev_ptrs_.size(), hipMemcpyHostToDevice);
        cudaCheckError();

        // copy offset
        hipMalloc((void **)&offset_device,
                sizeof(int64_t) * offset_list_.size());
        hipMemcpy(offset_device, &offset_list_[0],
                sizeof(int64_t) * offset_list_.size(),
                hipMemcpyHostToDevice);
        cudaCheckError();

        hipMalloc((void **)&access_book_device,
                sizeof(int) * access_book.size());
        hipMemcpy(access_book_device, &access_book[0],
                sizeof(int) * access_book.size(),
                hipMemcpyHostToDevice);
        cudaCheckError();

        int blockSize = 0;
        int numBlocks = 0;
        hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize,
                                           quiver_tensor_gather);
        // std::cout<<"LOG >>> "<<" numBlocks "<< numBlocks <<" blockSize
        // "<<blockSize<<std::endl;
        int ignore_access_book = 0;
        if(current_device != device_){
            ignore_access_book = 1;
        }
        quiver_tensor_gather<<<numBlocks, blockSize, 0, stream>>>(
            buffers_device, offset_device, offset_list_.size(),
            indices.data_ptr<int64_t>(), indices.numel(), res.data_ptr<float>(),
            stride(0), access_book_device, ignore_access_book);
        cudaCheckError();
        return res;
    }

    std::vector<int64_t> shape() const { return shape_; }

    int device() const { return device_; }

    int size(int dim) const { return shape_[dim]; }

    int64_t stride(int dim) const
    {
        int64_t res = 1;
        for (int index = dim + 1; index < shape_.size(); index++) {
            res *= shape_[index];
        }
        return res;
    }

    int64_t numel() const
    {
        int64_t res = 1;
        for (int index = 0; index < shape_.size(); index++) {
            res *= shape_[index];
        }
        return res;
    }
    std::vector<ShardTensorItem> share_ipc(){
        std::vector<ShardTensorItem> res;
        for(int index=0; index < dev_ptrs_.size(); index++){
            if(tensor_devices_[index] >= 0){
                hipSetDevice(tensor_devices_[index]);
                ShardTensorItem* item = new ShardTensorItem();
                item -> device = tensor_devices_[index];
                item -> shape = tensor_shapes_[index];
                hipIpcGetMemHandle(&(item -> mem_handle), dev_ptrs_[index]);
                res.push_back(*item);

            }
        }
        return res;
    }

    int device_count() const { return device_count_; }

    void unregister(torch::Tensor& cpu_tensor){

        std::cout<<"begin unregister"<<std::endl;
        hipHostUnregister((void*)cpu_tensor.data_ptr<float>());
        std::cout<<"end unregister"<<std::endl;

    }


  private:
    std::vector<int64_t> offset_list_;
    std::vector<float *> dev_ptrs_;
    std::vector<int> tensor_devices_;
    std::vector<int> access_book;
    std::vector<std::vector<int>> tensor_shapes_;
    std::vector<int64_t> shape_;
    int numa_broker_device;
    int device_;
    int device_count_;
    bool inited_;


};

void init_p2p(){
    std::cout << "LOG>>> P2P Access Initilization" << std::endl;
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    for (int i = 0; i < numGPUs; i++) {
        hipSetDevice(i);
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        // CUDA IPC is only supported on devices with unified addressing
        if (!prop.unifiedAddressing) {
            printf("Device %d does not support unified addressing, skipping...\n", i);
            continue;
        }
        // This sample requires two processes accessing each device, so we need
        // to ensure exclusive or prohibited mode is not set
        if (prop.computeMode != hipComputeModeDefault) {
            printf("Device %d is in an unsupported compute mode for this sample\n",
                i);
            continue;
        }

        for (int j = i + 1; j < numGPUs; j++) {
            if (can_device_access_peer(i, j)) {
                printf("Enable P2P Access Between %d <---> %d \n", i, j);
                hipSetDevice(i);
                hipDeviceEnablePeerAccess(j, 0);
                cudaCheckError();
                hipSetDevice(j);
                hipDeviceEnablePeerAccess(i, 0);
                cudaCheckError();
            }
        }
    }
}
}  // namespace quiver

void register_cuda_quiver_feature(pybind11::module &m)
{
    m.def("init_p2p", &quiver::init_p2p,
            py::call_guard<py::gil_scoped_release>());

    m.def("can_device_access_peer", &quiver::can_device_access_peer,
            py::call_guard<py::gil_scoped_release>());


    py::class_<quiver::ShardTensorItem>(m, "ShardTensorItem")
        .def(py::init<>())
        .def("share_ipc", &quiver::ShardTensorItem::share_ipc)
        .def("from_ipc", &quiver::ShardTensorItem::from_ipc);


    py::class_<quiver::ShardTensor>(m, "ShardTensor")
        //.def(py::init<std::vector<torch::Tensor>, int>())
        .def(py::init<int>())
        .def("__getitem__", &quiver::ShardTensor::operator[],
             py::call_guard<py::gil_scoped_release>())
        .def("unregister", &quiver::ShardTensor::unregister,
            py::call_guard<py::gil_scoped_release>())
        .def("shape", &quiver::ShardTensor::shape,
             py::call_guard<py::gil_scoped_release>())
        .def("numel", &quiver::ShardTensor::numel,
             py::call_guard<py::gil_scoped_release>())
        .def("device", &quiver::ShardTensor::device,
             py::call_guard<py::gil_scoped_release>())
        .def("stride", &quiver::ShardTensor::stride,
             py::call_guard<py::gil_scoped_release>())
        .def("size", &quiver::ShardTensor::size,
             py::call_guard<py::gil_scoped_release>())
        .def("device_count", &quiver::ShardTensor::device_count,
             py::call_guard<py::gil_scoped_release>())
        .def("append", py::overload_cast<torch::Tensor&, int>(&quiver::ShardTensor::append),
             py::call_guard<py::gil_scoped_release>())
        .def("append", py::overload_cast<quiver::ShardTensorItem>(&quiver::ShardTensor::append),
             py::call_guard<py::gil_scoped_release>())
        .def("share_ipc", &quiver::ShardTensor::share_ipc,
             py::call_guard<py::gil_scoped_release>());
}
