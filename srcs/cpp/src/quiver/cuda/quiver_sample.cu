#include <algorithm>
#include <numeric>

#include <thrust/device_vector.h>

#include <pybind11/numpy.h>
#include <torch/extension.h>

#include <quiver/common.hpp>
#include <quiver/functor.cu.hpp>
#include <quiver/quiver.cu.hpp>
#include <quiver/reindex.cu.hpp>
#include <quiver/stream_pool.hpp>
#include <quiver/trace.hpp>
#include <quiver/zip.hpp>
#include <thrust/remove.h>

template <typename IdType>
HostOrderedHashTable<IdType> *
FillWithDuplicates(const IdType *const input, const size_t num_input,
                   hipStream_t stream,
                   thrust::device_vector<IdType> &unique_items)
{
    const auto policy = thrust::cuda::par.on(stream);
    const int64_t num_tiles = (num_input + TILE_SIZE - 1) / TILE_SIZE;

    const dim3 grid(num_tiles);
    const dim3 block(BLOCK_SIZE);

    auto host_table = new HostOrderedHashTable<IdType>(num_input, 1);
    DeviceOrderedHashTable<IdType> device_table = host_table->DeviceHandle();

    generate_hashmap_duplicates<IdType, BLOCK_SIZE, TILE_SIZE>
        <<<grid, block, 0, stream>>>(input, num_input, device_table);
    thrust::device_vector<int> item_prefix(num_input + 1, 0);

    using it = thrust::counting_iterator<IdType>;
    using Mapping = typename DeviceOrderedHashTable<IdType>::Mapping;
    thrust::for_each(it(0), it(num_input),
                     [count = thrust::raw_pointer_cast(item_prefix.data()),
                      table = device_table,
                      in = input] __device__(IdType i) mutable {
                         Mapping &mapping = *(table.Search(in[i]));
                         if (mapping.index == i) { count[i] = 1; }
                     });
    thrust::exclusive_scan(item_prefix.begin(), item_prefix.end(),
                           item_prefix.begin());
    size_t tot = item_prefix[num_input];
    unique_items.resize(tot);

    thrust::for_each(it(0), it(num_input),
                     [prefix = thrust::raw_pointer_cast(item_prefix.data()),
                      table = device_table, in = input,
                      u = thrust::raw_pointer_cast(
                          unique_items.data())] __device__(IdType i) mutable {
                         Mapping &mapping = *(table.Search(in[i]));
                         if (mapping.index == i) {
                             mapping.local = prefix[i];
                             u[prefix[i]] = in[i];
                         }
                     });
    return host_table;
}

namespace quiver
{
template <typename T>
void replicate_fill(size_t n, const T *counts, const T *values, T *outputs)
{
    for (size_t i = 0; i < n; ++i) {
        const size_t c = counts[i];
        std::fill(outputs, outputs + c, values[i]);
        outputs += c;
    }
}

class TorchQuiver
{
    using torch_quiver_t = quiver<int64_t, CUDA>;
    torch_quiver_t quiver_;
    stream_pool pool_;

  public:
    TorchQuiver(torch_quiver_t quiver, int device = 0, int num_workers = 4)
        : quiver_(std::move(quiver))
    {
        pool_ = stream_pool(num_workers);
    }

    using T = int64_t;
    using W = float;

    // deprecated, not compatible with AliGraph
    std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
    sample_sub(const torch::Tensor &vertices, int k) const
    {
        return sample_sub_with_stream(0, vertices, k);
    }

    std::tuple<torch::Tensor, torch::Tensor>
    sample_neighbor(int stream_num, const torch::Tensor &vertices, int k)
    {
        hipStream_t stream = 0;
        if (!pool_.empty()) { stream = (pool_)[stream_num]; }
        const auto policy = thrust::cuda::par.on(stream);
        const size_t bs = vertices.size(0);
        thrust::device_vector<T> inputs;
        thrust::device_vector<T> outputs;
        thrust::device_vector<T> output_counts;
        sample_kernel(stream, vertices, k, inputs, outputs, output_counts);
        torch::Tensor neighbors =
            torch::empty(outputs.size(), vertices.options());
        torch::Tensor counts =
            torch::empty(vertices.size(0), vertices.options());
        thrust::copy(outputs.begin(), outputs.end(), neighbors.data_ptr<T>());
        thrust::copy(output_counts.begin(), output_counts.end(),
                     counts.data_ptr<T>());
        return std::make_tuple(neighbors, counts);
    }

    std::tuple<torch::Tensor, torch::Tensor>
    sample_kernel(const hipStream_t stream, const torch::Tensor &vertices,
                  int k, thrust::device_vector<T> &inputs,
                  thrust::device_vector<T> &outputs,
                  thrust::device_vector<T> &output_counts) const
    {
        T tot = 0;
        const auto policy = thrust::cuda::par.on(stream);
        thrust::device_vector<T> output_ptr;
        thrust::device_vector<T> output_idx;
        const T *p = vertices.data_ptr<T>();
        const size_t bs = vertices.size(0);

        {
            TRACE_SCOPE("alloc_1");
            inputs.resize(bs);
            output_counts.resize(bs);
            output_ptr.resize(bs);
        }
        // output_ptr is exclusive prefix sum of output_counts(neighbor counts
        // <= k)
        {
            TRACE_SCOPE("prepare");
            thrust::copy(p, p + bs, inputs.begin());
            // quiver_.to_local(stream, inputs);
            quiver_.degree(stream, inputs.data(), inputs.data() + inputs.size(),
                           output_counts.data());
            if (k >= 0) {
                thrust::transform(policy, output_counts.begin(),
                                  output_counts.end(), output_counts.begin(),
                                  cap_by<T>(k));
            }
            thrust::exclusive_scan(policy, output_counts.begin(),
                                   output_counts.end(), output_ptr.begin());
            tot = thrust::reduce(policy, output_counts.begin(),
                                 output_counts.end());
        }
        {
            TRACE_SCOPE("alloc_2");
            outputs.resize(tot);
            output_idx.resize(tot);
        }
        // outputs[outptr[i], outptr[i + 1]) are unique neighbors of inputs[i]
        // {
        //     TRACE_SCOPE("sample");
        //     quiver_.sample(stream, inputs.begin(), inputs.end(),
        //                    output_ptr.begin(), output_counts.begin(),
        //                    outputs.data(), output_eid.data());
        // }
        {
            TRACE_SCOPE("sample");
            quiver_.new_sample(
                stream, k, thrust::raw_pointer_cast(inputs.data()),
                inputs.size(), thrust::raw_pointer_cast(output_ptr.data()),
                thrust::raw_pointer_cast(output_counts.data()),
                thrust::raw_pointer_cast(outputs.data()),
                thrust::raw_pointer_cast(output_idx.data()));
        }
        torch::Tensor out_neighbor;
        torch::Tensor out_eid;

        // thrust::copy(outputs.begin(), outputs.end(),
        //              out_neighbor.data_ptr<T>());
        // thrust::copy(output_eid.begin(), output_eid.end(),
        //              out_eid.data_ptr<T>());
        return std::make_tuple(out_neighbor, out_eid);
    }

    static void reindex_kernel(const hipStream_t stream,
                               thrust::device_vector<T> &inputs,
                               thrust::device_vector<T> &outputs,
                               thrust::device_vector<T> &subset)
    {
        const auto policy = thrust::cuda::par.on(stream);
        HostOrderedHashTable<T> *table;
        // reindex
        {
            {
                TRACE_SCOPE("reindex 0");
                subset.resize(inputs.size() + outputs.size());
                thrust::copy(policy, inputs.begin(), inputs.end(),
                             subset.begin());
                thrust::copy(policy, outputs.begin(), outputs.end(),
                             subset.begin() + inputs.size());
                thrust::device_vector<T> unique_items;
                unique_items.clear();
                table =
                    FillWithDuplicates(thrust::raw_pointer_cast(subset.data()),
                                       subset.size(), stream, unique_items);
                subset.resize(unique_items.size());
                thrust::copy(policy, unique_items.begin(), unique_items.end(),
                             subset.begin());
                // thrust::sort(policy, subset.begin(), subset.end());
                // subset.erase(
                //     thrust::unique(policy, subset.begin(), subset.end()),
                //     subset.end());
                // _reindex_with(policy, outputs, subset, outputs);
            }
            {
                TRACE_SCOPE("permute");
                // thrust::device_vector<T> s1;
                // s1.reserve(subset.size());
                // _reindex_with(policy, inputs, subset, s1);
                // complete_permutation(s1, subset.size(), stream);
                // subset = permute(s1, subset, stream);

                // thrust::device_vector<T> s2;
                // inverse_permutation(s1, s2, stream);
                // permute_value(s2, outputs, stream);
                DeviceOrderedHashTable<T> device_table = table->DeviceHandle();
                thrust::for_each(
                    policy, outputs.begin(), outputs.end(),
                    [device_table] __device__(T & id) mutable {
                        using Iterator =
                            typename DeviceOrderedHashTable<T>::Iterator;
                        Iterator iter = device_table.Search(id);
                        id = static_cast<T>((*iter).local);
                    });
            }
            delete table;
        }
    }

    std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
    sample_sub_with_stream(int stream_num, const torch::Tensor &vertices,
                           int k) const
    {
        TRACE_SCOPE(__func__);
        hipStream_t stream = 0;
        if (!pool_.empty()) { stream = (pool_)[stream_num]; }
        const auto policy = thrust::cuda::par.on(stream);
        thrust::device_vector<T> inputs;
        thrust::device_vector<T> outputs;
        thrust::device_vector<T> output_counts;
        thrust::device_vector<T> subset;
        sample_kernel(stream, vertices, k, inputs, outputs, output_counts);
        int tot = outputs.size();

        reindex_kernel(stream, inputs, outputs, subset);

        torch::Tensor out_vertices =
            torch::empty(subset.size(), vertices.options());
        torch::Tensor row_idx = torch::empty(tot, vertices.options());
        torch::Tensor col_idx = torch::empty(tot, vertices.options());
        {
            TRACE_SCOPE("prepare output");
            thrust::device_vector<T> prefix_count(output_counts.size());
            thrust::device_vector<T> seq(output_counts.size());
            thrust::sequence(policy, seq.begin(), seq.end());
            thrust::exclusive_scan(policy, output_counts.begin(),
                                   output_counts.end(), prefix_count.begin());

            const size_t m = inputs.size();
            using it = thrust::counting_iterator<T>;
            thrust::for_each(
                policy, it(0), it(m),
                [prefix = thrust::raw_pointer_cast(prefix_count.data()),
                 count = thrust::raw_pointer_cast(output_counts.data()),
                 in = thrust::raw_pointer_cast(seq.data()),
                 out = thrust::raw_pointer_cast(
                     row_idx.data_ptr<T>())] __device__(T i) {
                    for (int j = 0; j < count[i]; j++) {
                        out[prefix[i] + j] = in[i];
                    }
                });
            thrust::copy(subset.begin(), subset.end(),
                         out_vertices.data_ptr<T>());
            thrust::copy(outputs.begin(), outputs.end(), col_idx.data_ptr<T>());
        }
        return std::make_tuple(out_vertices, row_idx, col_idx);
    }
};

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
reindex_single(torch::Tensor inputs, torch::Tensor outputs, torch::Tensor count)
{
    using T = int64_t;
    hipStream_t stream = 0;
    const auto policy = thrust::cuda::par.on(stream);
    thrust::device_vector<T> total_inputs(inputs.size(0));
    thrust::device_vector<T> total_outputs(outputs.size(0));
    thrust::device_vector<T> input_prefix(inputs.size(0));
    const T *ptr;
    size_t bs;
    ptr = count.data_ptr<T>();
    bs = inputs.size(0);
    thrust::copy(ptr, ptr + bs, input_prefix.begin());
    ptr = inputs.data_ptr<T>();
    thrust::copy(ptr, ptr + bs, total_inputs.begin());
    thrust::exclusive_scan(policy, input_prefix.begin(), input_prefix.end(),
                           input_prefix.begin());
    ptr = outputs.data_ptr<T>();
    bs = outputs.size(0);
    thrust::copy(ptr, ptr + bs, total_outputs.begin());

    const size_t m = inputs.size(0);
    using it = thrust::counting_iterator<T>;

    thrust::device_vector<T> subset;
    TorchQuiver::reindex_kernel(stream, total_inputs, total_outputs, subset);

    int tot = total_outputs.size();
    torch::Tensor out_vertices = torch::empty(subset.size(), inputs.options());
    torch::Tensor row_idx = torch::empty(tot, inputs.options());
    torch::Tensor col_idx = torch::empty(tot, inputs.options());
    {
        thrust::device_vector<T> seq(count.size(0));
        thrust::sequence(policy, seq.begin(), seq.end());

        thrust::for_each(
            policy, it(0), it(m),
            [prefix = thrust::raw_pointer_cast(input_prefix.data()),
             count = count.data_ptr<T>(),
             in = thrust::raw_pointer_cast(seq.data()),
             out = thrust::raw_pointer_cast(
                 row_idx.data_ptr<T>())] __device__(T i) {
                for (int j = 0; j < count[i]; j++) {
                    out[prefix[i] + j] = in[i];
                }
            });
        thrust::copy(subset.begin(), subset.end(), out_vertices.data_ptr<T>());
        thrust::copy(total_outputs.begin(), total_outputs.end(),
                     col_idx.data_ptr<T>());
    }
    return std::make_tuple(out_vertices, row_idx, col_idx);
}

TorchQuiver new_quiver_from_csr_array(torch::Tensor &input_indptr,
                                      torch::Tensor &input_indices,
                                      torch::Tensor &input_edge_idx,
                                      int device = 0, bool cuda = false)
{

    hipSetDevice(device);
    TRACE_SCOPE(__func__);

    using T = typename TorchQuiver::T;

    check_eq<int64_t>(input_indptr.dim(), 1);
    const size_t node_count = input_indptr.size(0);

    check_eq<int64_t>(input_indices.dim(), 1);
    const size_t edge_count = input_indices.size(0);

    bool use_eid = input_edge_idx.size(0) == edge_count;


    /*
    In Zero-Copy Mode, We Do These Steps:
    0. Copy The Data If Needed
    1. Register Buffer As Mapped Pinned Memory
    2. Get Device Pointer In GPU Memory Space
    3. Intiliaze A Quiver Instance And Return
    */

    T *indptr_device_pointer = nullptr;
    T *indices_device_pointer = nullptr;
    T *edge_id_device_pointer = nullptr;
    {
        /*if (!cuda) {
            const T *indptr_original = reinterpret_cast<const T *>(input_indptr.data_ptr<T>());
            // Register Buffer As Mapped Pinned Memory
            quiverRegister((void *)indptr_original, sizeof(T) * node_count,
                             hipHostRegisterMapped);
            // Get Device Pointer In GPU Memory Space
            hipHostGetDevicePointer((void **)&indptr_device_pointer,
                                     (void *)indptr_original, 0);
        } else */{
            const T *indptr_original = reinterpret_cast<const T *>(input_indptr.data_ptr<T>());
            T *indptr_copy;
            hipMalloc((void **)&indptr_copy, sizeof(T) * node_count);
            hipMemcpy((void *)indptr_copy, (void *)indptr_original,
                   sizeof(T) * node_count, hipMemcpyDefault);
            indptr_device_pointer = indptr_copy;
        }

    }
    // std::cout<<"mapped indptr"<<std::endl;
    {
        if (!cuda) {
            const T *indices_original =
                reinterpret_cast<const T *>(input_indices.data_ptr<T>());
            // Register Buffer As Mapped Pinned Memory
            quiverRegister((void *)indices_original, sizeof(T) * edge_count,
                             hipHostRegisterMapped);
            // Get Device Pointer In GPU Memory Space
            hipHostGetDevicePointer((void **)&indices_device_pointer,
                                     (void *)indices_original, 0);
        } else {
            const T *indices_original =
                reinterpret_cast<const T *>(input_indices.data_ptr<T>());
            T *indices_copy;
            hipMalloc((void **)&indices_copy, sizeof(T) * edge_count);
            hipMemcpy((void *)indices_copy, (void *)indices_original,
                   sizeof(T) * edge_count, hipMemcpyDefault);
            indices_device_pointer = indices_copy;
        }

    }

    // std::cout<<"mapped indices"<<std::endl;
    if (use_eid) {
        if (!cuda) {
            const T *id_original = reinterpret_cast<const T *>(input_edge_idx.data_ptr<T>());
            // Register Buffer As Mapped Pinned Memory
            quiverRegister((void *)id_original, sizeof(T) * edge_count,
                             hipHostRegisterMapped);
            // Get Device Pointer In GPU Memory Space
            hipHostGetDevicePointer((void **)&edge_id_device_pointer,
                                     (void *)id_original, 0);
        } else {
            const T *id_original = reinterpret_cast<const T *>(input_edge_idx.data_ptr<T>());
            T *id_copy;
            hipMalloc((void **)&id_copy, sizeof(T) * edge_count);
            hipMemcpy((void *)id_copy, (void *)id_original,
                   sizeof(T) * edge_count, hipMemcpyDefault);
            edge_id_device_pointer = id_copy;
        }
    }

    // std::cout<<"mapped edge id "<<std::endl;
    // initialize Quiver instance
    using Q = quiver<int64_t, CUDA>;
    Q quiver = Q::New(indptr_device_pointer, indices_device_pointer,
                      edge_id_device_pointer, node_count-1, edge_count);
    return TorchQuiver(std::move(quiver), device);
}

TorchQuiver new_quiver_from_edge_index(size_t n,
                                       py::array_t<int64_t> &input_edges,
                                       py::array_t<int64_t> &input_edge_idx,
                                       int device = 0)
{
    hipSetDevice(device);
    TRACE_SCOPE(__func__);
    using T = typename TorchQuiver::T;
    py::buffer_info edges = input_edges.request();
    py::buffer_info edge_idx = input_edge_idx.request();
    check_eq<int64_t>(edges.ndim, 2);
    check_eq<int64_t>(edges.shape[0], 2);
    const size_t m = edges.shape[1];
    check_eq<int64_t>(edge_idx.ndim, 1);

    bool use_eid = edge_idx.shape[0] == m;

    thrust::device_vector<T> row_idx(m);
    thrust::device_vector<T> col_idx(m);
    {
        const T *p = reinterpret_cast<const T *>(edges.ptr);
        thrust::copy(p, p + m, row_idx.begin());
        thrust::copy(p + m, p + m * 2, col_idx.begin());
    }
    thrust::device_vector<T> edge_idx_;
    if (use_eid) {
        edge_idx_.resize(m);
        const T *p = reinterpret_cast<const T *>(edge_idx.ptr);
        thrust::copy(p, p + m, edge_idx_.begin());
    }
    using Q = quiver<int64_t, CUDA>;
    Q quiver = Q::New(static_cast<T>(n), std::move(row_idx), std::move(col_idx),
                      std::move(edge_idx_));
    return TorchQuiver(std::move(quiver), device);
}
}  // namespace quiver

void register_cuda_quiver_sample(pybind11::module &m)
{
    m.def("reindex_single", &quiver::reindex_single);
    m.def("new_quiver_from_edge_index", &quiver::new_quiver_from_edge_index);
    m.def("new_quiver_from_csr_array", &quiver::new_quiver_from_csr_array);
    py::class_<quiver::TorchQuiver>(m, "Quiver")
        .def("sample_sub", &quiver::TorchQuiver::sample_sub_with_stream,
             py::call_guard<py::gil_scoped_release>())
        .def("sample_neighbor", &quiver::TorchQuiver::sample_neighbor,
             py::call_guard<py::gil_scoped_release>());
}
